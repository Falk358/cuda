
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <fstream>
using namespace std;


__global__
void k_matvecmul(long n, double* in, double* out, double* mat) {
    long i = threadIdx.x + blockDim.x*blockIdx.x;

    if(i>0 && i < n-1)
        out[i] = mat[0]*in[i] + mat[1]*in[i+1] + mat[2]*in[i-1];
}


int main() {
    long n = 1e6;
    long time_steps = 100;
    double *h_in, *d_mat;
    double *d_in, *d_out;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // To be more realistic, we do not time memory allocation.
    hipHostMalloc(&h_in, sizeof(double)*n, hipHostMallocDefault);
    hipMalloc(&d_in,  sizeof(double)*n);
    hipMalloc(&d_out, sizeof(double)*n);

    hipEventRecord(start, 0);

    // intialize vector and copy to GPU
    for(long i=0;i<n;i++)
        h_in[i] = sin(double(i)/double(n-1)*M_PI);

    hipMemcpy(d_in, h_in, sizeof(double)*n, hipMemcpyHostToDevice);

    // initialize matrix and copy to GPU
    double matrix_row[3] = {1.0 - 0.25*2.0, 0.25, 0.25};
    hipMalloc(&d_mat, sizeof(double)*3);
    hipMemcpy(d_mat, matrix_row, sizeof(double)*3, hipMemcpyHostToDevice);

    // repeated matrix-vector multiplication (i.e. time integration)
    for(long k=0;k<time_steps;k++) {
        k_matvecmul<<<n/128+1,128>>>(n, d_in, d_out, d_mat);

        hipMemcpy(d_in, d_out, sizeof(double)*n, hipMemcpyDeviceToDevice);
    }

    hipEventRecord(stop, 0);

    // Write result to a file (we do not time this).
    hipMemcpy(h_in, d_in, sizeof(double)*n, hipMemcpyDeviceToHost);
    ofstream fs("result.data");
    for(long i=0;i<n;i++)
        fs << h_in[i] << endl;
    fs.close();

    // Compare to the exact solution (we do not time this).
    double error = 0.0;
    double decay = exp(-0.25/pow(double(n-1),2)*time_steps*pow(M_PI,2));
    for(long i=0;i<n;i++)
        error = max(error, fabs(h_in[i]- decay*sin(double(i)/double(n-1)*M_PI)));
    cout << "Numerical error: " << error << endl;

    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    cout << time*1e-3 << " s" << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}