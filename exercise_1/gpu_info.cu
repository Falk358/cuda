
#include <hip/hip_runtime.h>
#include <stdio.h>
int main()
{
	int deviceCount;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);
		printf("\nDevice %d: %s", i, deviceProp.name);	
	}


}
