
#include <hip/hip_runtime.h>
#include <stdio.h>
int main()
{
	int deviceCount;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipDeviceProp_t deviceProp;
		int driverVersion;
		int runtimeVersion;
		hipError_t error_id_driver = hipDriverGetVersion(&driverVersion);
		hipError_t error_id_runtime = hipRuntimeGetVersion(&runtimeVersion);
		hipGetDeviceProperties(&deviceProp, i);
		printf("\nDevice %d: %s\n", i, deviceProp.name);
		printf("	Driver Version %d\n 	Runtime Version: %d\n 	GPU Memory in MB: %d\n 	Max Num Threads per block: %d\n 	UVA available: %d", driverVersion, runtimeVersion, deviceProp.totalGlobalMem/1024/1024, deviceProp.maxThreadsPerBlock, deviceProp.unifiedAddressing);
		
	}


}

