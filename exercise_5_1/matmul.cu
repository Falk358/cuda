
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <hipblas.h>
using namespace std;


// init n*n matrix filled with value val
double* initMatrix(long n, double val, size_t memsize)
{
    double* matrix = (double*) malloc(memsize);
    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < n; i++)
        {
            matrix[i + j*n] = val;
        }
    }
    return matrix;
}

double* initVector(long n, double val, size_t memsize)
{
    double* vector = (double*) malloc(memsize);

    for (int i = 0; i < n; i++)
    {
        vector[i] = val;
    }
    
    return vector;
}



int verifyResult(double* result, long n_result, double tolerance)
{
    return;
}



int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        cerr << "Error, not enough cmd line arguments given; usage matmul \"problem_size\", where problem size is an int";
        exit(1);
    }
    int problem_size = stoi(argv[1]);
    size_t vec_size = sizeof(double) *problem_size;
    size_t mat_size = sizeof(double) * problem_size* problem_size;
    double* vector = initVector(problem_size, 1.0, vec_size);
    double* matrix = initMatrix(problem_size, 2.0, mat_size);
    double* result = (double*) malloc(mat_size);
    
    double alpha = 1.0;
    double beta = 0.0;
    double* device_vector;
    double* device_matrix;
    double* device_result;
    hipMalloc(&device_vector, vec_size);
    hipMalloc(&device_matrix, mat_size);
    hipMalloc(&device_result, mat_size);

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        cout << "ERROR: hipblasCreate failed. Error code: " << status << endl;
        exit(1);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start,0);

    //TODO computation here
    hipblasStatus_t status_computation = hipblasDgemv(handle,HIPBLAS_OP_N, problem_size, problem_size, &alpha, device_matrix,1,device_vector, 1, &beta, device_result,1);
    if (status_computation != HIPBLAS_STATUS_SUCCESS)
    {
        cout << "ERROR: hipblasDgemv failed. Error code" << status_computation << endl;
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    cout << time*1e-3 << " s" << endl;


    // free memory
    hipblasDestroy(handle); 
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(device_vector);
    hipFree(device_matrix);
    hipFree(device_result);
    free(matrix);    
    free(vector);
    free(result);


}