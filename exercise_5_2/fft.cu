
#include <hip/hip_runtime.h>
#include <math.h>
#include <complex>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <iostream>


#define FUNCTION_RESOLUTION 10
#define BLOCK_SIZE 128
#define THREAD_COUNT 128


// implements f(x) = sin(2 * pi * x) for values 0.1 .. 1.0
double* initSine(size_t array_size, double* x_points) 
{
    hipfftDoubleReal* y_points = (hipfftDoubleReal*) malloc(array_size);
    size_t x_length = sizeof(x_points) / sizeof(x_points[0]);
    for (int i = 0; i < x_length; i++)
    {
        y_points[i] = sin(M_PI*2*x_points[i]);
    }
    return y_points;

}

// verfiy with analytical solution f''(x) = 1/(2*pi)² sin(2*pi*x)
bool verifyResult(hipfftDoubleReal* result, double tolerance, double* x_points)
{
    bool verified = true;
    size_t x_length = sizeof(x_points) / sizeof(x_points[0]);
    for (int i = 0; i< x_length; i++)
    {
        double analytical_solution = (1/pow(2.0 * M_PI,2.0)) * sin(2.0*M_PI*x_points[i]);
        double error = fabs(analytical_solution - result[i]);
        if (error > tolerance)
        {
            verified = false;
            std::cerr << "Error verifying at index " << i << "\n error: " << error << "\n analytical solution: " << analytical_solution << "\n calculated solution: " << result[i] << std::endl;
        }
    }
    return verified;
    
}

// implents u_k = f_k / (2*pi*k)²
__global__
void calcInComplexSpace(hipfftDoubleComplex* f_k, hipfftDoubleComplex* u_k, size_t n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > n)
    {
        return;
    }
    else if (i == 0)
    {
        hipfftDoubleComplex at_0;
        at_0.x = 0.0;
        at_0.y = 0.0;
        u_k[0] = at_0;
    }
    else
    {
        u_k[i].x = f_k[i].x / pow((2 * M_PI * i), 2.0);
        u_k[i].y = f_k[i].y / pow((2 * M_PI * i), 2.0);
    }
    return;
}





int main()
{
    size_t array_size = sizeof(hipfftDoubleReal) * FUNCTION_RESOLUTION;
    size_t array_size_freq = ((FUNCTION_RESOLUTION/ 2) + 1) * sizeof(hipfftDoubleComplex);
    hipfftDoubleReal* y_points;
    hipfftDoubleReal* u_points;
    hipfftDoubleReal* device_y_points;
    hipfftDoubleReal* device_u_points;
    hipfftDoubleComplex* device_y_points_freq;
    hipfftDoubleComplex* device_u_points_freq;

    double x_points[FUNCTION_RESOLUTION] = {0.1, 0.2, 0.3, 0.4 , 0.5 ,0.6, 0.7, 0.8, 0.9, 1.0};
    y_points = initSine(array_size, x_points);
    u_points = (hipfftDoubleReal*) malloc(array_size);
    hipMalloc(&device_y_points, array_size);
    hipMalloc(&device_u_points, array_size);
    hipMalloc(&device_y_points_freq, array_size_freq);
    hipMalloc(&device_u_points_freq, array_size_freq);
    hipMemcpy(device_y_points, y_points, array_size, hipMemcpyHostToDevice);
    
    hipfftHandle fft_forward_handle;
    
    hipfftHandle fft_backward_handle;

    hipfftResult plan_forward_success = hipfftPlan1d(&fft_forward_handle, array_size, HIPFFT_D2Z, 1);
    if (plan_forward_success  != HIPFFT_SUCCESS)
    {
        std::cerr << "Error creating plan with array_size " << array_size << " with error: " << plan_forward_success << std::endl; 
    }
    hipfftResult plan_backward_success = hipfftPlan1d(&fft_backward_handle, array_size, HIPFFT_Z2D, 1);
    if (plan_backward_success != HIPFFT_SUCCESS)
    {
        std::cerr << "Error creating plan with array_size " << array_size << " with error: " << plan_backward_success << std::endl; 
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // TODO FFT here
    hipfftResult res_forward = hipfftExecD2Z(fft_forward_handle, device_y_points, device_y_points_freq);
    if (res_forward != HIPFFT_SUCCESS)
    {
        std::cerr << "Error computing frequency domain of y points. Message: " << res_forward << std::endl;
    }
    calcInComplexSpace<<<BLOCK_SIZE, THREAD_COUNT>>>(device_y_points_freq, device_u_points_freq, array_size_freq);
    // TODO convert u back to space domain (backward)
    hipfftResult res_backward = hipfftExecZ2D(fft_backward_handle, device_u_points_freq, device_u_points);
    if (res_backward != HIPFFT_SUCCESS)
    {
        std::cerr << "Error computing space domain from frequency domain of u points. Message: " << res_backward << std::endl;
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    std::cout << time*1e-3 << " s" << std::endl;
    hipMemcpy(u_points, device_u_points, array_size, hipMemcpyDeviceToHost);

    bool verified = verifyResult(u_points, 0.1, x_points);
    if (verified)
    {
        std::cout << "SUCCESS!!! result was verified with analytical solution!!!" << std::endl;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipfftDestroy(plan_forward_success);
    hipfftDestroy(plan_backward_success);
    

    free(y_points);
    free(u_points);
    hipFree(device_u_points);
    hipFree(device_y_points);
    hipFree(device_y_points_freq);
    hipFree(device_u_points_freq);
}